
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_complex.h"

#define I2PI (1<<16)
#define MASK (I2PI-1)

__global__ void initarraygpu(hipFloatComplex[], int);

__global__ void computestrucf(hipFloatComplex*, hipFloatComplex*,
			      int, int,
			      int, int, int);

extern "C"{
  void cudastrucf_(hipFloatComplex *csf, hipFloatComplex *cex, float *xat, int *nxat, int *num, float *xm, float *win, float *vin, float *uin, int *cr_natoms)
  {
    int nnum = num[0]*num[1]*num[2];
    
    int threadsPerBlock = 64;
    int threadsPerGrid = (nnum + threadsPerBlock - 1) / threadsPerBlock;
    
    hipFloatComplex* d_tcsf;
    hipMalloc((void**) &d_tcsf, nnum * sizeof(hipFloatComplex));
    
    hipFloatComplex* d_cex;
    hipMalloc((void**) &d_cex, I2PI * sizeof(hipFloatComplex));
    
    hipMemcpy(d_cex, cex, I2PI * sizeof(hipFloatComplex), hipMemcpyHostToDevice);
    
    initarraygpu<<<threadsPerGrid, threadsPerBlock>>>(d_tcsf, nnum);
    
    printf("Starting CUDA!!!!\n");
    
    float xarg0, xincu, xincv;//, xincw;
    int iarg0, iincu, iincv;//, iincw;
    
    for(int l=0; l< nxat[0]; l++){
      xarg0 = xm[0] * xat[l] + xm[1] * xat[l+cr_natoms[0]] + xm[2] * xat[l+cr_natoms[0]*2];
      xincu = uin[0] * xat[l] + uin[1] * xat[l+cr_natoms[0]] + uin[2] * xat[l+cr_natoms[0]*2];
      xincv = vin[0] * xat[l] + vin[1] * xat[l+cr_natoms[0]] + vin[2] * xat[l+cr_natoms[0]*2];
      //xincw = win1 * xat1 + win2 * xat2 + win3 * xat3;
      iarg0 = (int)rintf(64 * I2PI * (xarg0 - (int)xarg0 + 1.));
      iincu = (int)rintf(64 * I2PI * (xincu - (int)xincu + 1.));
      iincv = (int)rintf(64 * I2PI * (xincv - (int)xincv + 1.));
      //iincw = (int)rintf(64 * I2PI * (xincw - (int)xincw + 1.));
      
      computestrucf<<<threadsPerGrid, threadsPerBlock>>>
	(d_cex, d_tcsf,
	 num[0],num[1],
	 iarg0,iincu,iincv);
    }
    
    hipMemcpy(csf, d_tcsf, nnum*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    hipMemcpy(csf, d_tcsf, nnum*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
        
    hipFree(d_tcsf);
    hipFree(d_cex);
    
  }
}

__global__ void computestrucf(hipFloatComplex* cex, hipFloatComplex* tcsf,
			      int num1, int num2,
			      int iarg0, int iincu, int iincv)
{
  int i, j, iadd, id, iarg;
  
  id = threadIdx.x + blockDim.x * blockIdx.x;
  if(id<num1*num2)
    {
      i = id / num1;
      j = id % num1;
      iarg = iarg0 + i * iincu + j * iincv;
      iadd = iarg >> 6;
      iadd = iadd & MASK;
      tcsf[id] = hipCaddf(tcsf[id],cex[iadd]);
    };
  __syncthreads();
}


__global__ void initarraygpu(hipFloatComplex* array1, int nelements)
{
  int id = threadIdx.x + blockDim.x * blockIdx.x;
  if(id<nelements)
    {
      array1[id] = make_hipFloatComplex(0.0,0.0);
    };
  __syncthreads();
}

